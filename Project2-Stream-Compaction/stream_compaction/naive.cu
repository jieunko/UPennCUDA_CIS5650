#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <iostream>

namespace StreamCompaction {
    namespace Naive 
    {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void naiveParallelScan(int n, int* odata, const int* idata, int level) 
        {
            int index =  blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) return;
      
            int levelOffset = 1 << (level - 1);
            int valueToAdd = (index >= levelOffset) ? idata[index - levelOffset] : 0;
            odata[index] = valueToAdd + idata[index];
        }

            /**
             * Performs prefix-sum (aka scan) on idata, storing the result into odata.
             */
        void scan(int n, int* odata, const int* idata)
        {
            int* dev_a;
            int* dev_b;
            hipMalloc((void**)&dev_a, n * sizeof(int));
            checkCUDAError("hipMalloc dev_a failed!");
            hipMalloc((void**)&dev_b, n * sizeof(int));
            checkCUDAError("hipMalloc dev_b failed!");
            hipMemcpy(dev_a, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 256;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int numLevels = ilog2ceil(n);

            timer().startGpuTimer();
            for (int i = 1; i <= numLevels; ++i)
            {
                naiveParallelScan<<<fullBlocksPerGrid, blockSize >>>(n, dev_b, dev_a, i);
                std::swap(dev_a, dev_b);
            }

            timer().endGpuTimer();

            //hipDeviceSynchronize();
            odata[0] = 0;
            hipMemcpy(odata+1, dev_a, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_a);
            hipFree(dev_b);
        }
    }
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void upSweep(int n, int* inplacedata, int level)
        {
            int index =  blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) return;
            
            int frontOffset = 1 << (level+1);
            int backOffset = (1 << level);
            int bound = frontOffset - 1;
            if((index%frontOffset) == bound) inplacedata[index] += inplacedata[index - backOffset];
        }

        __global__ void downSweep(int n, int* inplacedata, int level)
        {
            int index =  blockIdx.x * blockDim.x + threadIdx.x;
            if (index >= n) return;
      
            int frontOffset = 1 << (level+1);
            int backOffset = 1 << level;
            int bound = frontOffset - 1;
            if (index % frontOffset == bound)
            {
                int temp = inplacedata[index - backOffset];
                inplacedata[index - backOffset] = inplacedata[index];
                inplacedata[index] += temp;
            }
        }
        


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            int* dev_idata;
            int numLevels = ilog2ceil(n);
            int extended = pow(2, numLevels );
            int arrSize = n % 2 == 0 ? n : extended;
            hipMalloc((void**)&dev_idata, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 256;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            

            timer().startGpuTimer();
            // TODO
            for (int d = 0; d < numLevels; ++d)
            {
                upSweep<<<fullBlocksPerGrid, blockSize >>>(n, dev_idata, d);
            }
            
            hipMemset(&dev_idata[arrSize - 1], 0, sizeof(int));


            for (int d = numLevels - 1; d >= 0; --d)
            {
                downSweep<<<fullBlocksPerGrid, blockSize >>>(arrSize, dev_idata, d);
            }
            
            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_idata);


        }

        void indeviceScan(int n, int numLevels,  int* odata, const int* idata)
        {
            int* dev_idata;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy Device to Device idata  failed!");

            int blockSize = 256;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            
            // TODO     
            for (int d = 0; d < numLevels; ++d)
            {
                upSweep<<<fullBlocksPerGrid, blockSize >>>(n, dev_idata, d);
                checkCUDAError("upSweep %d failed!", d);
            }
            
            hipMemset(&dev_idata[n - 1], 0, sizeof(int));


            for (int d = numLevels - 1; d >= 0; --d)
            {
                downSweep<<<fullBlocksPerGrid, blockSize >>>(n, dev_idata, d);
                checkCUDAError("downSweep %d failed!", d);
            }

            hipMemcpy(odata, dev_idata, n* sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy Device to Device dev_idata  failed!");
            hipFree(dev_idata);
        }

        


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_bools;
            int* dev_indicies;
            int* dev_scatter;
            int numLevels = ilog2ceil(n);
            int extended = pow(2, numLevels );
            int arrSize = n % 2 == 0 ? n : extended;
            int t;
            int t2;
         
            hipMalloc((void**)&dev_idata, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_bools, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_indicies, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_indicies failed!");
            hipMalloc((void**)&dev_scatter, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_scatter failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(&dev_idata[n], 0, (arrSize-n) * sizeof(int));

            int blockSize = 256;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // TODO
            Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(arrSize, dev_bools, dev_idata);
            checkCUDAError("kernMapToBoolean failed!");
            indeviceScan(arrSize,numLevels, dev_indicies, dev_bools);
            
            Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(arrSize, dev_scatter, dev_idata, dev_bools, dev_indicies);
            checkCUDAError("kernScatter failed!");
            timer().endGpuTimer();
            

            
            hipMemcpy(&t,  &dev_indicies[arrSize-1], sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&t2, &dev_bools[arrSize-1], sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_indicies failed!");

            hipMemcpy(odata, dev_scatter, (t+t2) * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed!");

            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_indicies);
            hipFree(dev_scatter);
            return t;
        }
    }
}
